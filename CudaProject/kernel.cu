﻿#include "../Matrix/Matrix.h"
#include "../MatrixOperations/MatrixOperations.cuh"
#include "../LinearRegression/LinearRegression.cuh"


int main() {
	// Test linear regression
	Matrix X_lin = Matrix::read_csv("C:/Users/Dmitry/source/repos/lewinskydmitry/CUDA/CudaProject/data_linear/X.csv",',');
	Matrix y_lin = Matrix::read_csv("C:/Users/Dmitry/source/repos/lewinskydmitry/CUDA/CudaProject/data_linear/y.csv", ',');

	LinearRegression reg;
	Matrix C_lin = reg.fit(X_lin, y_lin, 30);

	reg.losses.print();

    return 0;
}