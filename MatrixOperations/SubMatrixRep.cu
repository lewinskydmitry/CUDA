#include "hip/hip_runtime.h"
#include "MatrixOperations.cuh"


// Kernel for performing matrices substraction with replacement
__global__ void SubMatrixRepKernel(Matrix& A, Matrix B)
{
    int size = A.width * A.length;
    int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_idx < size) {
        A.data[thread_idx] -= B.data[thread_idx];
    }
}


// Host code for performing matrices substraction with replacement
void SubMatrixRep(Matrix& A, Matrix B) {

    // This code for catching errors if dimensions of matrices don't match
    if (A.length != B.length && A.width != B.width) {
        try {
            throw std::invalid_argument("Dimensions do not match");
        }
        catch (const std::invalid_argument& e) {
            std::cout << "Matrix addition error:" << "\n";
            std::cout << e.what() << std::endl;
            exit(1);
        }
    }

    Matrix d_A;
    d_A.width = A.width; d_A.length = A.length;
    int size = A.width * A.length;
    hipMalloc(&d_A.data, size * sizeof(double));
    hipMemcpy(d_A.data, A.data, size * sizeof(double), hipMemcpyHostToDevice);

    Matrix d_B;
    d_B.width = B.width; d_B.length = B.length;
    hipMalloc(&d_B.data, size * sizeof(double));
    hipMemcpy(d_B.data, B.data, size * sizeof(double), hipMemcpyHostToDevice);

    int blocksPerGrid = (d_A.width * d_A.length + threadsPerBlock - 1) / threadsPerBlock;
    SubMatrixRepKernel << < blocksPerGrid, threadsPerBlock >> > (d_A, d_B);

    hipMemcpy(A.data, d_A.data, size * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_A.data);
    hipFree(d_B.data);
};