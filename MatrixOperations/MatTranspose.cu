#include "hip/hip_runtime.h"
#include "MatrixOperations.cuh"

__global__ void copySharedMem(Matrix odata, const Matrix idata)
{
    __shared__ double tile[BLOCK_SIZE * BLOCK_SIZE];

    int x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int y = blockIdx.y * BLOCK_SIZE + threadIdx.y;

    for (int j = 0; j < BLOCK_SIZE; j += BLOCK_SIZE)
        tile[(threadIdx.y + j) * BLOCK_SIZE + threadIdx.x] = idata.data[(y + j) * idata.width + x];

    __syncthreads();

    for (int j = 0; j < BLOCK_SIZE; j += BLOCK_SIZE)
        odata.data[(y + j) * odata.width + x] = tile[(threadIdx.x + j) * BLOCK_SIZE + threadIdx.y];
}


Matrix MatTranspose(Matrix A)
{
    Matrix d_A;
    d_A.width = A.width; d_A.length = A.length;
    size_t size = A.width * A.length;
    hipMalloc(&d_A.data, size * sizeof(double));
    hipMemcpy(d_A.data, A.data, size * sizeof(double), hipMemcpyHostToDevice);

    Matrix d_C;
    d_C.length = A.width;
    d_C.width = A.length;
    size = A.length * A.width;
    hipMalloc(&d_C.data, size * sizeof(double));

    // Invoke kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((A.width + dimBlock.x - 1) / dimBlock.x, (A.length + dimBlock.y - 1) / dimBlock.y);
    copySharedMem << <dimGrid, dimBlock >> > (d_C, d_A);

    Matrix C;
    C.length = A.width;
    C.width = A.length;
    C.data = new double[size];

    // Read C from device memory
    hipMemcpy(C.data, d_C.data, size * sizeof(double), hipMemcpyDeviceToHost);
    // Free device memory
    hipFree(d_A.data);
    hipFree(d_C.data);
    return C;
}