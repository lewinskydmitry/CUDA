#include "hip/hip_runtime.h"
#include "MatrixOperations.cuh"


__global__ void TransposeKernelRep(Matrix idata) {
    __shared__ double tile[BLOCK_SIZE][BLOCK_SIZE];
    int x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int y = blockIdx.y * BLOCK_SIZE + threadIdx.y; 
    int i;

    for (int i = 0; i < BLOCK_SIZE; i += blockDim.y) {
        if (x < idata.width && (y + i) < idata.length) {
            tile[threadIdx.y + i][threadIdx.x] = idata.data[(y + i) * idata.width + x];
        }
    }

    __syncthreads();

    x = blockIdx.y * BLOCK_SIZE + threadIdx.x;
    y = blockIdx.x * BLOCK_SIZE + threadIdx.y;

    for (int i = 0; i < BLOCK_SIZE; i += blockDim.y) {
        if (x < idata.length && (y + i) < idata.width) {
            idata.data[(y + i) * idata.length + x] = tile[threadIdx.x][threadIdx.y + i];

        }
    }
}


void TransposeRep(Matrix& A)
{
    Matrix d_A;
    d_A.width = A.width; d_A.length = A.length;
    size_t size = A.width * A.length;
    hipMalloc(&d_A.data, size * sizeof(double));
    hipMemcpy(d_A.data, A.data, size * sizeof(double), hipMemcpyHostToDevice);

    // Invoke kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((A.width + dimBlock.x - 1) / dimBlock.x, (A.length + dimBlock.y - 1) / dimBlock.y);
    TransposeKernelRep <<< dimGrid, dimBlock >>> (d_A);

    hipMemcpy(A.data, d_A.data, size * sizeof(double), hipMemcpyDeviceToHost);

    A.length = d_A.width;
    A.width = d_A.length;

    hipFree(d_A.data);
}