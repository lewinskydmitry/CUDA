#include "hip/hip_runtime.h"
#include "MatrixOperations.cuh"

__global__ void TransposeKernelRep(Matrix Matrixdata)
{
    __shared__ double tile[BLOCK_SIZE * BLOCK_SIZE];

    int x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int y = blockIdx.y * BLOCK_SIZE + threadIdx.y;

    tile[threadIdx.y * BLOCK_SIZE + threadIdx.x] = Matrixdata.data[y * Matrixdata.width + x];
    __syncthreads();
    Matrixdata.data[y * Matrixdata.width + x] = tile[threadIdx.x * BLOCK_SIZE + threadIdx.y];
}


void TransposeRep(Matrix A)
{
    Matrix d_A;
    d_A.width = A.width; d_A.length = A.length;
    size_t size = A.width * A.length;
    hipMalloc(&d_A.data, size * sizeof(double));
    hipMemcpy(d_A.data, A.data, size * sizeof(double), hipMemcpyHostToDevice);

    // Invoke kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((A.width + dimBlock.x - 1) / dimBlock.x, (A.length + dimBlock.y - 1) / dimBlock.y);
    TransposeKernelRep << <dimGrid, dimBlock >> > (d_A);

    // Read C from device memory
    hipMemcpy(A.data, d_A.data, size * sizeof(double), hipMemcpyDeviceToHost);
    // Free device memory
    hipFree(d_A.data);
}