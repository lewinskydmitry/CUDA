#include "hip/hip_runtime.h"
#include "MatrixOperations.cuh"

__global__
void cudaAddMatrixKernel(const Matrix A,
    const Matrix B,
    Matrix C) {
    int size = A.width * A.length;
    int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (thread_idx < size) {
        C.data[thread_idx] = A.data[thread_idx] + B.data[thread_idx];
        thread_idx += blockDim.x * gridDim.x;
    }
}


Matrix AddMatrix(Matrix A, Matrix B) {

    Matrix d_A;
    d_A.width = A.width; d_A.length = A.length;
    size_t size = A.width * A.length;
    hipMalloc(&d_A.data, size * sizeof(double));
    hipMemcpy(d_A.data, A.data, size * sizeof(double), hipMemcpyHostToDevice);

    Matrix d_B;
    d_B.width = B.width; d_B.length = B.length;
    hipMalloc(&d_B.data, size * sizeof(double));
    hipMemcpy(d_B.data, B.data, size * sizeof(double), hipMemcpyHostToDevice);


    Matrix d_C;
    d_C.width = B.width; d_C.length = B.length;
    hipMalloc(&d_C.data, size * sizeof(double));
    hipMemcpy(d_C.data, B.data, size * sizeof(double), hipMemcpyHostToDevice);

    int per_block_thread_count = 1024;
    int block_count = (int)ceil(size / (int)per_block_thread_count);

    cudaAddMatrixKernel << < block_count, per_block_thread_count >> > (d_A, d_B, d_C);

    hipDeviceSynchronize();

    Matrix C;
    C.width = B.width; C.length = B.length;
    C.data = new double[C.width * C.length];

    hipMemcpy(C.data, d_C.data, size * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_A.data);
    hipFree(d_B.data);
    hipFree(d_C.data);
    return C;
};