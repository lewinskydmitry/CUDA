#include "hip/hip_runtime.h"
#include "MatrixOperations.cuh"


__global__
void cudaAddMatrixKernel(const Matrix A, const Matrix B, Matrix C) 
{
    int size = A.width * A.length;
    int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (thread_idx < size) {
        C.data[thread_idx] = A.data[thread_idx] + B.data[thread_idx];
        thread_idx += blockDim.x * gridDim.x;
    }
}


Matrix AddMatrix(Matrix A, Matrix B) {

    if (A.length != B.length && A.width != B.width) {
        try {
            throw std::invalid_argument("Dimensions do not match");
        }
        catch (const std::invalid_argument& e) {
            std::cout << "Matrix addition error:" << "\n";
            std::cout << e.what() << std::endl;
            exit(1);
        }
    }

    Matrix d_A;
    d_A.width = A.width; d_A.length = A.length;
    int size = A.width * A.length;
    hipMalloc(&d_A.data, size * sizeof(double));
    hipMemcpy(d_A.data, A.data, size * sizeof(double), hipMemcpyHostToDevice);

    Matrix d_B;
    d_B.width = B.width; d_B.length = B.length;
    hipMalloc(&d_B.data, size * sizeof(double));
    hipMemcpy(d_B.data, B.data, size * sizeof(double), hipMemcpyHostToDevice);


    Matrix d_C;
    d_C.width = B.width; d_C.length = B.length;
    hipMalloc(&d_C.data, size * sizeof(double));


    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((A.width + dimBlock.x - 1) / dimBlock.x, (A.length + dimBlock.y - 1) / dimBlock.y);
    
    cudaAddMatrixKernel << < dimGrid, dimBlock >> > (d_A, d_B, d_C);

    Matrix C;
    C.width = B.width; C.length = B.length;
    C.data = new double[C.width * C.length];

    hipMemcpy(C.data, d_C.data, size * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_A.data);
    hipFree(d_B.data);
    hipFree(d_C.data);

    return C;
};