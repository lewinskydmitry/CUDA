#include "hip/hip_runtime.h"
#include "vectorSum.cuh"

__global__
void cudaAddVectorKernel(const double* a,
    const double* b,
    double* c,
    const int size) {
    int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (thread_idx < size) {
        c[thread_idx] = a[thread_idx] + b[thread_idx];
        thread_idx += blockDim.x * gridDim.x;
    }
}

void cudaCallAddVectorKernel(const double* a,
    const double* b,
    double* c,
    const int size) {

    double* dev_a = 0;
    double* dev_b = 0;
    double* dev_c = 0;
    int per_block_thread_count = 1024;

    int block_count = (int)ceil(size / (float)per_block_thread_count);


    hipMalloc((void**)&dev_c, size * sizeof(double));
    hipMalloc((void**)&dev_a, size * sizeof(double));
    hipMalloc((void**)&dev_b, size * sizeof(double));


    hipMemcpy(dev_a, a, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(double), hipMemcpyHostToDevice);

    cudaAddVectorKernel <<< block_count, per_block_thread_count >>> (dev_a, dev_b, dev_c, size);

    hipDeviceSynchronize();

    hipMemcpy(c, dev_c, size * sizeof(double), hipMemcpyDeviceToHost);


    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
};